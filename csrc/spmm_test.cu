/***************************************************************************
 * Copyright 2025 The SpInfer Authors. All rights reserved.
 * Copyright 2023 The FLash-LLM Authors. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ***************************************************************************/
#include "./SpMM_API.cuh"

#include "./spmm_test_utils.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#define TTTTT 0
int main(int argc, char **argv) {

    // int M_GLOBAL = 8192;
    // int K_GLOBAL = 8192;
    // int N_GLOBAL = 1024;

    int M_GLOBAL = 36864;
    int K_GLOBAL = 36864;
    int N_GLOBAL = 36864;
    int MATRIX_A_PRUNING_PERCENTAGE = 50;
    int SPLIT_K = 1;
    hipblasStatus_t cublas_status;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Host memory
    half *A_h = NULL; // row major
    half *B_h = NULL; // col major
    // Device memory
    half *A = NULL;
    half *B = NULL;
    A_h = (half *)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    B_h = (half *)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
   
    if (A_h == NULL || B_h == NULL) {
        printf("Error in CPU Malloc!\n");
        exit(-1);
    }
    hipMalloc(reinterpret_cast<void **>(&A), sizeof(half) * M_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void **>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);

    checkLastCudaError(__LINE__);
    if (A == NULL || B == NULL) {
        printf("Error in hipMalloc!\n");
        exit(-1);
    }
    //

    init_host_matrices(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);

    printf("Preparing dense data for GPU...\n");
    hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);

    // CUBLAS
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuBlas...\n");
    half *D_cublas = NULL;
    hipMalloc(reinterpret_cast<void **>(&D_cublas), sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cublas == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_cublas, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, 0);

    int m = M_GLOBAL, n = N_GLOBAL, k = K_GLOBAL;
    const float alpha = 1.0;
    const float beta = 0.0;
    hipblasGemmAlgo_t CuBlasALG = static_cast<hipblasGemmAlgo_t>(0);
    // Tensor core enabled
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    hipDeviceSynchronize();
    for (int i = 0; i < WARM_UP_ITERATION; i++) {
        cublas_status = hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, HIP_R_16F, k, B, HIP_R_16F, k, &beta, D_cublas,
                                     HIP_R_16F, m, HIP_R_32F, CuBlasALG);
        checkCublasError(cublas_status, __LINE__);
    }
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, HIP_R_16F, k, B, HIP_R_16F, k, &beta, D_cublas, HIP_R_16F, m,
                     HIP_R_32F, CuBlasALG);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    //
    float milliseconds_cublas_tc = 0;
    hipEventElapsedTime(&milliseconds_cublas_tc, start, stop);
    milliseconds_cublas_tc = milliseconds_cublas_tc / BENCHMARK_ITERATION;
    float tflops_cublas_tc = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_cublas_tc / 1000.)) / 1e12;
    half *D_cublas_h = NULL; // col major
    D_cublas_h = (half *)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_cublas_h == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_cublas_h, D_cublas, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost); // Col Major
    hipFree(D_cublas);
    /////////////////////////////////////////////////////////////////////////////////////////////////

    auto Split_K = SPLIT_K;

    // SpInfer
    ////////////////////////////////////////////////////////////////////////////////////////////////
    half *D_SpMM_bitmapv3 = NULL;
    hipMalloc(reinterpret_cast<void **>(&D_SpMM_bitmapv3), sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_SpMM_bitmapv3 == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_SpMM_bitmapv3, 0, sizeof(half) * M_GLOBAL * N_GLOBAL);

    // Define the output pointer
    half *Compressed_Val_cpu_v3 = nullptr;
    int *bitmap_TileOffsets_cpu_v3 = nullptr;
    int *bitmap_TileOffsets_median_cpu_v3 = nullptr;
    int *bitmap_TileOffsets_global_cpu_v3 = nullptr;
    uint64_t *bitmap_cpu_v3 = nullptr;
    int max_nnz_intilev3 = 0;
    // Call the InitSparseMatrixA_bitmap_v6 function
    auto num_gtilesv3 =
        InitSparseMatrixA_bitmap_v6(A_h, M_GLOBAL, K_GLOBAL, 8, 16, 64, 8, 64, 64, &Compressed_Val_cpu_v3, &bitmap_TileOffsets_cpu_v3,
                                    &bitmap_TileOffsets_median_cpu_v3, &bitmap_TileOffsets_global_cpu_v3, &bitmap_cpu_v3, max_nnz_intilev3);

//     print_bitmap_v3_results(Compressed_Val_cpu_v3, bitmap_TileOffsets_cpu_v3, bitmap_TileOffsets_global_cpu_v3, bitmap_cpu_v3, num_gtilesv3,
//                             num_gtilesv3, max_nnz_intilev3);

    auto local_tile_numv3 = 8 * 8;
    auto median_tile_numv3 = 4 * 1;
    auto num_ltilesv3 = num_gtilesv3 * local_tile_numv3;
    auto num_mtilesv3 = num_gtilesv3 * median_tile_numv3;
    // The offset of the last tile is equal to the total number of compressed non-zero values
    int val_count_v3 = bitmap_TileOffsets_global_cpu_v3[num_gtilesv3];
    int val_count_median_v3 = bitmap_TileOffsets_median_cpu_v3[num_mtilesv3];
    // Adjust max_nnz_intilev3 to a multiple of 64
    if (max_nnz_intilev3 % 64 != 0) {
        max_nnz_intilev3 = ((max_nnz_intilev3 / 64) + 1) * 64;
    }
    printf("num_global_tiles: %d, bitmap v3 NNZ: %d, bitmap v3 median layer NNZ: %d,  max_nnz_intilev3: %d \n", num_gtilesv3, val_count_v3,
           val_count_median_v3, max_nnz_intilev3);
    half *Compressed_Val_gpu_v3 = nullptr;
    int *bitmap_TileOffsets_gpu_v3 = nullptr;
    int *bitmap_TileOffsets_median_gpu_v3 = nullptr;
    int *bitmap_TileOffsets_global_gpu_v3 = nullptr;
    uint64_t *bitmap_gpu_v3 = nullptr;
    hipMalloc(&bitmap_TileOffsets_gpu_v3, sizeof(int) * (num_ltilesv3 + 1)); // for (16*64 tile specific)
    hipMalloc(&bitmap_gpu_v3, sizeof(uint64_t) * (num_ltilesv3));
    hipMalloc(&bitmap_TileOffsets_median_gpu_v3, sizeof(int) * (num_mtilesv3));
    hipMalloc(&bitmap_TileOffsets_global_gpu_v3, sizeof(int) * (num_gtilesv3 + 1));
    if (val_count_v3 == 0)
        val_count_v3 = 1; // For 100% sparsity, NNZ = 0, malloc will return NULL
    hipMalloc(&Compressed_Val_gpu_v3, sizeof(half) * val_count_v3);
    if (bitmap_TileOffsets_gpu_v3 == NULL || bitmap_gpu_v3 == NULL || Compressed_Val_gpu_v3 == NULL || bitmap_TileOffsets_global_gpu_v3 == NULL) {
        printf("Error in malloc memory from device memory!\n");
        exit(-1);
    }
    hipMemcpy(bitmap_TileOffsets_gpu_v3, bitmap_TileOffsets_cpu_v3, sizeof(int) * (num_ltilesv3 + 1), hipMemcpyHostToDevice);
    hipMemcpy(bitmap_TileOffsets_global_gpu_v3, bitmap_TileOffsets_global_cpu_v3, sizeof(int) * (num_gtilesv3 + 1), hipMemcpyHostToDevice);
    hipMemcpy(bitmap_TileOffsets_median_gpu_v3, bitmap_TileOffsets_median_cpu_v3, sizeof(int) * (num_mtilesv3), hipMemcpyHostToDevice);
    hipMemcpy(bitmap_gpu_v3, bitmap_cpu_v3, sizeof(uint64_t) * num_ltilesv3, hipMemcpyHostToDevice);
    hipMemcpy(Compressed_Val_gpu_v3, Compressed_Val_cpu_v3, sizeof(half) * val_count_v3, hipMemcpyHostToDevice);
    free(bitmap_TileOffsets_cpu_v3);
    free(bitmap_cpu_v3);
    free(Compressed_Val_cpu_v3);
    free(bitmap_TileOffsets_global_cpu_v3);
    free(bitmap_TileOffsets_median_cpu_v3);
    printf("Done! Compressed A matrix for bitmap v3 GPU kernel.\n");

    // Compress B matrix similar to A
    printf("Compressing B matrix...\n");
    half *Compressed_B_cpu_v3 = nullptr;
    int *B_bitmap_TileOffsets_cpu_v3 = nullptr;
    int *B_bitmap_TileOffsets_median_cpu_v3 = nullptr;
    int *B_bitmap_TileOffsets_global_cpu_v3 = nullptr;
    uint64_t *B_bitmap_cpu_v3 = nullptr;
    int B_max_nnz_intilev3 = 0;

    // Call the InitSparseMatrixA_bitmap_v6 function for B (notice B is K_GLOBAL x N_GLOBAL, in column-major order)
    auto B_num_gtilesv3 =
        InitSparseMatrixA_bitmap_v6_B(B_h, K_GLOBAL, N_GLOBAL, 8, 16, 64, 8, 16, 64, &Compressed_B_cpu_v3, &B_bitmap_TileOffsets_cpu_v3,
                                      &B_bitmap_TileOffsets_median_cpu_v3, &B_bitmap_TileOffsets_global_cpu_v3, &B_bitmap_cpu_v3, B_max_nnz_intilev3);
    auto B_local_tile_numv3 = 8 * 8;
    auto B_median_tile_numv3 = 4 * 1;
    auto B_num_ltilesv3 = B_num_gtilesv3 * B_local_tile_numv3;
    auto B_num_mtilesv3 = B_num_gtilesv3 * B_median_tile_numv3;

    // The offset of the last tile is equal to the total number of compressed non-zero values
    int B_val_count_v3 = B_bitmap_TileOffsets_global_cpu_v3[B_num_gtilesv3];
    int B_val_count_median_v3 = B_bitmap_TileOffsets_median_cpu_v3[B_num_mtilesv3];

    // Adjust B_max_nnz_intilev3 to a multiple of 64
    if (B_max_nnz_intilev3 % 64 != 0) {
        B_max_nnz_intilev3 = ((B_max_nnz_intilev3 / 64) + 1) * 64;
    }

    printf("B num_global_tiles: %d, bitmap v3 NNZ: %d, bitmap v3 median layer NNZ: %d, max_nnz_intilev3: %d \n", B_num_gtilesv3, B_val_count_v3,
           B_val_count_median_v3, B_max_nnz_intilev3);

    // Allocate device memory for compressed B
    half *Compressed_B_gpu_v3 = nullptr;
    int *B_bitmap_TileOffsets_gpu_v3 = nullptr;
    int *B_bitmap_TileOffsets_median_gpu_v3 = nullptr;
    int *B_bitmap_TileOffsets_global_gpu_v3 = nullptr;
    uint64_t *B_bitmap_gpu_v3 = nullptr;

    hipMalloc(&B_bitmap_TileOffsets_gpu_v3, sizeof(int) * (B_num_ltilesv3 + 1));
    hipMalloc(&B_bitmap_gpu_v3, sizeof(uint64_t) * (B_num_ltilesv3));
    hipMalloc(&B_bitmap_TileOffsets_median_gpu_v3, sizeof(int) * (B_num_mtilesv3));
    hipMalloc(&B_bitmap_TileOffsets_global_gpu_v3, sizeof(int) * (B_num_gtilesv3 + 1));

    if (B_val_count_v3 == 0)
        B_val_count_v3 = 1; // For 100% sparsity, NNZ = 0, malloc will return NULL

    hipMalloc(&Compressed_B_gpu_v3, sizeof(half) * B_val_count_v3);

    if (B_bitmap_TileOffsets_gpu_v3 == NULL || B_bitmap_gpu_v3 == NULL || Compressed_B_gpu_v3 == NULL || B_bitmap_TileOffsets_global_gpu_v3 == NULL) {
        printf("Error in malloc memory from device memory for compressed B!\n");
        exit(-1);
    }

    // Copy compressed B data to device
    hipMemcpy(B_bitmap_TileOffsets_gpu_v3, B_bitmap_TileOffsets_cpu_v3, sizeof(int) * (B_num_ltilesv3 + 1), hipMemcpyHostToDevice);
    hipMemcpy(B_bitmap_TileOffsets_global_gpu_v3, B_bitmap_TileOffsets_global_cpu_v3, sizeof(int) * (B_num_gtilesv3 + 1), hipMemcpyHostToDevice);
    hipMemcpy(B_bitmap_TileOffsets_median_gpu_v3, B_bitmap_TileOffsets_median_cpu_v3, sizeof(int) * (B_num_mtilesv3), hipMemcpyHostToDevice);
    hipMemcpy(B_bitmap_gpu_v3, B_bitmap_cpu_v3, sizeof(uint64_t) * B_num_ltilesv3, hipMemcpyHostToDevice);
    hipMemcpy(Compressed_B_gpu_v3, Compressed_B_cpu_v3, sizeof(half) * B_val_count_v3, hipMemcpyHostToDevice);

    // Free CPU memory for compressed B
    free(B_bitmap_TileOffsets_cpu_v3);
    free(B_bitmap_cpu_v3);
    free(Compressed_B_cpu_v3);
    free(B_bitmap_TileOffsets_global_cpu_v3);
    free(B_bitmap_TileOffsets_median_cpu_v3);

    printf("Done! Compressed B matrix for bitmap v3 GPU kernel.\n");

    printf("Launching bitmapv3 without Ahead of Time Sparse Data Reordering...\n");
    Split_K = SPLIT_K;
    printf("Split_K = %d\n", Split_K);
    half *Reduction_Workspace_bitmapv3 = NULL;
    hipMalloc(reinterpret_cast<void **>(&Reduction_Workspace_bitmapv3), sizeof(half) * M_GLOBAL * N_GLOBAL * Split_K);
    if (Reduction_Workspace_bitmapv3 == NULL) {
        printf("Error in hipMalloc\n");
        exit(-1);
    }
    int *max_nnz_intilev3_gpu = nullptr;
    hipMalloc(&max_nnz_intilev3_gpu, sizeof(int));
    if (max_nnz_intilev3_gpu == NULL) {
        printf("Error in hipMalloc for max_nnz_intilev3_gpu\n");
        exit(-1);
    }
    hipMemcpy(max_nnz_intilev3_gpu, &max_nnz_intilev3, sizeof(int), hipMemcpyHostToDevice);

    int *B_max_nnz_intilev3_gpu = nullptr;
    hipMalloc(&B_max_nnz_intilev3_gpu, sizeof(int));
    if (B_max_nnz_intilev3_gpu == NULL) {
        printf("Error in hipMalloc for B_max_nnz_intilev3_gpu\n");
        exit(-1);
    }
    hipMemcpy(B_max_nnz_intilev3_gpu, &B_max_nnz_intilev3, sizeof(int), hipMemcpyHostToDevice);

    for (int i = 0; i < WARM_UP_ITERATION; i++)
        SpMM_SplitK_API_bitmap_v3(0, A,
                                  Compressed_Val_gpu_v3,            // half
                                  bitmap_TileOffsets_global_gpu_v3, // int
                                  bitmap_TileOffsets_median_gpu_v3, // int
                                  bitmap_gpu_v3,                    // uint64
                                  max_nnz_intilev3_gpu,             // int
                                  B, Compressed_B_gpu_v3, B_bitmap_TileOffsets_global_gpu_v3, B_bitmap_TileOffsets_median_gpu_v3, B_bitmap_gpu_v3,
                                  B_max_nnz_intilev3_gpu, D_SpMM_bitmapv3, M_GLOBAL, N_GLOBAL, K_GLOBAL, Reduction_Workspace_bitmapv3, Split_K);
    hipEventRecord(start);
    for (int i = 0; i < BENCHMARK_ITERATION; i++)
        SpMM_SplitK_API_bitmap_v3(0, A, Compressed_Val_gpu_v3, bitmap_TileOffsets_global_gpu_v3, bitmap_TileOffsets_median_gpu_v3, bitmap_gpu_v3,
                                  max_nnz_intilev3_gpu, B, Compressed_B_gpu_v3, B_bitmap_TileOffsets_global_gpu_v3,
                                  B_bitmap_TileOffsets_median_gpu_v3, B_bitmap_gpu_v3, B_max_nnz_intilev3_gpu, D_SpMM_bitmapv3, M_GLOBAL, N_GLOBAL,
                                  K_GLOBAL, Reduction_Workspace_bitmapv3, Split_K);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    checkLastCudaError(__LINE__);
    // //
    float milliseconds_SpMM_bitmapv3 = 0.0f;
    hipEventElapsedTime(&milliseconds_SpMM_bitmapv3, start, stop);
    milliseconds_SpMM_bitmapv3 = milliseconds_SpMM_bitmapv3 / BENCHMARK_ITERATION;
    float tflops_SpMM_bitmapv3 =
        static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2) / (milliseconds_SpMM_bitmapv3 / 1000.)) / 1e12;
    half *D_SpMM_hbitmapv3 = NULL; // col major
    D_SpMM_hbitmapv3 = (half *)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    hipMemcpy(D_SpMM_hbitmapv3, D_SpMM_bitmapv3, sizeof(half) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost); // Col Major
    hipFree(D_SpMM_bitmapv3);
    hipFree(bitmap_TileOffsets_gpu_v3);
    hipFree(bitmap_TileOffsets_global_gpu_v3);
    hipFree(bitmap_TileOffsets_median_gpu_v3);
    hipFree(bitmap_gpu_v3);
    hipFree(Compressed_Val_gpu_v3);
    hipFree(Reduction_Workspace_bitmapv3);
    hipFree(max_nnz_intilev3_gpu);

    double totalError_SpMM_bitmapv3 = 0.0;

    totalError_SpMM_bitmapv3 = ComputeTotalError(D_cublas_h, D_SpMM_hbitmapv3, M_GLOBAL, N_GLOBAL);

    free(D_SpMM_hbitmapv3);

    PrintPerformance("SpInfer", milliseconds_SpMM_bitmapv3, tflops_SpMM_bitmapv3, totalError_SpMM_bitmapv3);
    PrintPerformance("CuBlas_TC", milliseconds_cublas_tc, tflops_cublas_tc, 0.0);

    free(D_cublas_h);
    free(A_h);
    free(B_h);

    hipFree(A);
    hipFree(B);

    return 0;
}
